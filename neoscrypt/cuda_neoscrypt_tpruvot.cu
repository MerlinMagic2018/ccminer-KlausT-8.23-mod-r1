#include "hip/hip_runtime.h"
﻿// originally from djm34 - github.com/djm34/ccminer-sp-neoscrypt
// kernel code from Nanashi Meiyo-Meijin 1.7.6-r10 (July 2016)
// modified by tpruvot

#include <stdio.h>
#include <memory.h>
#include "cuda_helper.h"
#include "cuda_vector_uint2x4.cuh"
#include "cuda_vector_tpruvot.cuh" 
#include "miner.h"

#ifdef _MSC_VER
#define THREAD __declspec(thread)
#else
#define THREAD __thread
#endif

#define rotate ROTL32
#define rotateR ROTR32
#define rotateL ROTL32

#ifdef __INTELLISENSE__
/* just for vstudio code colors */
#if __CUDA_ARCH__ >= 320
__device__ uint32_t __funnelshift_l(uint32_t lo, uint32_t hi, uint32_t shift);
__device__ ​uint32_t __funnelshift_r(uint32_t lo, uint32_t hi, uint32_t shift);
#endif
#endif
#if __CUDA_ARCH__ < 320
#define __funnelshift_l(lo, hi, shift) (((lo) >> (32 - (shift))) | ((hi) << (shift)))
#define __funnelshift_r(lo, hi, shift) (((hi) << (32 - (shift))) | ((lo) >> (shift)))
#define __ldg(x) (*(x))
#define __ldg4(x) (*(x))
#endif
#if defined(CUDART_VERSION) && CUDART_VERSION < 9000
#define __syncwarp(mask) __threadfence_block()
#endif

typedef uint48 uint4x2;

static uint32_t* d_NNonce[MAX_GPUS];

__device__ uint2x4* W;
__device__ uint2x4* Tr;
__device__ uint2x4* Tr2;
__device__ uint2x4* Input;

__constant__ uint32_t c_data[64];
__constant__ uint32_t c_target[2];
__constant__ uint32_t key_init[16];
__constant__ uint32_t input_init[16];

static const __constant__ uint8 BLAKE2S_IV_Vec = {
	0x6A09E667, 0xBB67AE85, 0x3C6EF372, 0xA54FF53A,
	0x510E527F, 0x9B05688C, 0x1F83D9AB, 0x5BE0CD19
};

static const uint8 BLAKE2S_IV_Vechost = {
	0x6A09E667, 0xBB67AE85, 0x3C6EF372, 0xA54FF53A,
	0x510E527F, 0x9B05688C, 0x1F83D9AB, 0x5BE0CD19
};

static const uint32_t BLAKE2S_SIGMA_host[10][16] = {
	{0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15},
	{14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3},
	{11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4},
	{7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8},
	{9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13},
	{2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9},
	{12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11},
	{13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10},
	{6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5},
	{10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0},
};

__constant__ uint32_t BLAKE2S_SIGMA[10][16] = {
	{0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15},
	{14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3},
	{11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4},
	{7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8},
	{9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13},
	{2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9},
	{12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11},
	{13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10},
	{6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5},
	{10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0},
};

#define BLOCK_SIZE         64U
#define BLAKE2S_BLOCK_SIZE 64U
#define BLAKE2S_OUT_SIZE   32U

#define SALSA(a,b,c,d) { \
	t = rotateL(a + d,  7U); b ^= t; \
	t = rotateL(b + a,  9U); c ^= t; \
	t = rotateL(c + b, 13U); d ^= t; \
	t = rotateL(d + c, 18U); a ^= t; \
}

#if __CUDA_ARCH__ < 300
#define bitselect(a, b, c) ((a) ^ ((c) & ((b) ^ (a))))

__device__ __forceinline__ uint32_t WarpShuffle(uint32_t a, uint32_t b, uint32_t c)
{
	extern __shared__ uint32_t shared_mem[];
	uint32_t thread = threadIdx.y * blockDim.x + threadIdx.x;
	uint32_t threads = blockDim.y * blockDim.x;
	uint32_t buf, result;

	__syncwarp(0xFFFFFFFF);
	buf = shared_mem[threads * 0 + thread];
	shared_mem[threads * 0 + thread] = a;
	__syncwarp(0xFFFFFFFF);
	result = shared_mem[0 * threads + bitselect(threadIdx.x, b, c)];
	__syncwarp(0xFFFFFFFF);
	shared_mem[threads * 0 + thread] = buf;

	return result;
}

__device__ __forceinline__ void WarpShuffle3(uint32_t &d0, uint32_t &d1, uint32_t &d2, uint32_t a0, uint32_t a1, uint32_t a2, uint32_t b0, uint32_t b1, uint32_t b2, uint32_t c)
{
	extern __shared__ uint32_t shared_mem[];
	uint32_t thread = threadIdx.y * blockDim.x + threadIdx.x;
	uint32_t threads = blockDim.y * blockDim.x;
	uint32_t buf0, buf1, buf2;

	__syncwarp(0xFFFFFFFF);
	buf0 = shared_mem[threads * 0 + thread];
	buf1 = shared_mem[threads * 1 + thread];
	buf2 = shared_mem[threads * 2 + thread];
	shared_mem[threads * 0 + thread] = a0;
	shared_mem[threads * 1 + thread] = a1;
	shared_mem[threads * 2 + thread] = a2;
	__syncwarp(0xFFFFFFFF);
	d0 = shared_mem[0 * threads + bitselect(threadIdx.x, b0, c)];
	d1 = shared_mem[1 * threads + bitselect(threadIdx.x, b1, c)];
	d2 = shared_mem[2 * threads + bitselect(threadIdx.x, b2, c)];
	__syncwarp(0xFFFFFFFF);
	shared_mem[threads * 0 + thread] = buf0;
	shared_mem[threads * 1 + thread] = buf1;
	shared_mem[threads * 2 + thread] = buf2;
}

#else
__device__ __forceinline__ uint32_t WarpShuffle(uint32_t a, uint32_t b, uint32_t c)
{
	return SHFL(a, b, c);
}

__device__ __forceinline__ void WarpShuffle3(uint32_t &d0, uint32_t &d1, uint32_t &d2, uint32_t a0, uint32_t a1, uint32_t a2, uint32_t b0, uint32_t b1, uint32_t b2, uint32_t c)
{
	d0 = WarpShuffle(a0, b0, c);
	d1 = WarpShuffle(a1, b1, c);
	d2 = WarpShuffle(a2, b2, c);
}

#endif

#define CHACHA_STEP(a,b,c,d) { \
	a += b; d = __byte_perm(d ^ a, 0, 0x1032); \
	c += d; b = rotateL(b ^ c, 12); \
	a += b; d = __byte_perm(d ^ a, 0, 0x2103); \
	c += d; b = rotateL(b ^ c, 7); \
}

#if __CUDA_ARCH__ < 500

__device__ __forceinline__
static void shift256R4(uint32_t* ret, const uint8 &vec4, const uint32_t shift2)
{
	uint32_t shift = 32U - shift2;

	ret[0] = vec4.s0 << (32 - shift);
	ret[1] = __funnelshift_r(vec4.s0, vec4.s1, shift);
	ret[2] = __funnelshift_r(vec4.s1, vec4.s2, shift);
	ret[3] = __funnelshift_r(vec4.s2, vec4.s3, shift);
	ret[4] = __funnelshift_r(vec4.s3, vec4.s4, shift);
	ret[5] = __funnelshift_r(vec4.s4, vec4.s5, shift);
	ret[6] = __funnelshift_r(vec4.s5, vec4.s6, shift);
	ret[7] = __funnelshift_r(vec4.s6, vec4.s7, shift);
	ret[8] = vec4.s7 >> shift;
}

#define BLAKE(a, b, c, d, key1, key2) { \
	a += key1; \
	a += b; d = rotateL(d ^ a, 16); \
	c += d; b = rotateR(b ^ c, 12); \
	a += key2; \
	a += b; d = rotateR(d ^ a, 8); \
	c += d; b = rotateR(b ^ c, 7); \
}

#define BLAKE_G(idx0, idx1, a, b, c, d, key) { \
	idx = BLAKE2S_SIGMA[idx0][idx1]; a += key[idx]; \
	a += b; d = rotate(d ^ a, 16); \
	c += d; b = rotateR(b ^ c, 12); \
	idx = BLAKE2S_SIGMA[idx0][idx1 + 1]; a += key[idx]; \
	a += b; d = rotateR(d ^ a, 8); \
	c += d; b = rotateR(b ^ c, 7); \
}

#define BLAKE_G_PRE(idx0, idx1, a, b, c, d, key) { \
	a += key[idx0]; \
	a += b; d = rotateL(d ^ a, 16); \
	c += d; b = rotateR(b ^ c, 12); \
	a += key[idx1]; \
	a += b; d = rotateR(d ^ a, 8); \
	c += d; b = rotateR(b ^ c, 7); \
}

#define BLAKE_G_PRE0(idx0, idx1, a, b, c, d, key) { \
	a += b; d = rotateL(d ^ a, 16); \
	c += d; b = rotateR(b ^ c, 12); \
	a += b; d = rotateR(d ^ a, 8); \
	c += d; b = rotateR(b ^ c, 7); \
}

#define BLAKE_G_PRE1(idx0, idx1, a, b, c, d, key) { \
	a += key[idx0]; \
	a += b; d = rotateL(d ^ a, 16); \
	c += d; b = rotateR(b ^ c, 12); \
	a += b; d = rotateR(d ^ a, 8); \
	c += d; b = rotateR(b ^ c, 7); \
}

#define BLAKE_G_PRE2(idx0, idx1, a, b, c, d, key) { \
	a += b; d = rotateL(d ^ a, 16); \
	c += d; b = rotateR(b ^ c, 12); \
	a += key[idx1]; \
	a += b; d = rotateR(d ^ a, 8); \
	c += d; b = rotateR(b ^ c, 7); \
}

static __forceinline__ __device__
void Blake2S(uint32_t *out, const uint32_t* const __restrict__  inout, const  uint32_t * const __restrict__ TheKey)
{
	uint16 V;
	uint32_t idx;
	uint8 tmpblock;

	V.hi = BLAKE2S_IV_Vec;
	V.lo = BLAKE2S_IV_Vec;
	V.lo.s0 ^= 0x01012020;

	// Copy input block for later
	tmpblock = V.lo;

	V.hi.s4 ^= BLAKE2S_BLOCK_SIZE;

	//	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	BLAKE_G_PRE(0, 1, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(2, 3, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(4, 5, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE(6, 7, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE0(8, 9, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE0(10, 11, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE0(12, 13, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE0(14, 15, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	// { 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	BLAKE_G_PRE0(14, 10, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE1(4, 8, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE0(9, 15, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE2(13, 6, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE1(1, 12, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(0, 2, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE2(11, 7, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE(5, 3, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	// { 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	BLAKE_G_PRE0(11, 8, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE2(12, 0, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(5, 2, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE0(15, 13, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE0(10, 14, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(3, 6, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(7, 1, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE2(9, 4, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	// { 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	BLAKE_G_PRE1(7, 9, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(3, 1, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE0(13, 12, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE0(11, 14, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(2, 6, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE1(5, 10, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(4, 0, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE0(15, 8, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	// { 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	BLAKE_G_PRE2(9, 0, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(5, 7, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(2, 4, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE0(10, 15, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE2(14, 1, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE0(11, 12, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE1(6, 8, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE1(3, 13, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	// { 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
	BLAKE_G_PRE1(2, 12, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE1(6, 10, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE1(0, 11, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE2(8, 3, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE1(4, 13, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(7, 5, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE0(15, 14, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE1(1, 9, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	// { 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
	BLAKE_G_PRE2(12, 5, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE1(1, 15, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE0(14, 13, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE1(4, 10, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(0, 7, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(6, 3, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE2(9, 2, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE0(8, 11, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	// { 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
	BLAKE_G_PRE0(13, 11, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE1(7, 14, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE2(12, 1, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE1(3, 9, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(5, 0, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE2(15, 4, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE2(8, 6, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE(2, 10, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	// { 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
	BLAKE_G_PRE1(6, 15, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE0(14, 9, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE2(11, 3, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE1(0, 8, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE2(12, 2, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE2(13, 7, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(1, 4, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE2(10, 5, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	// { 10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0 },
	BLAKE_G_PRE2(10, 2, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE2(8, 4, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(7, 6, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE(1, 5, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE0(15, 11, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE0(9, 14, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE1(3, 12, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE2(13, 0, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);

	V.lo ^= V.hi ^ tmpblock;

	V.hi = BLAKE2S_IV_Vec;
	tmpblock = V.lo;

	V.hi.s4 ^= 128;
	V.hi.s6 = ~V.hi.s6;

	// { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	BLAKE_G_PRE(0, 1, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
	BLAKE_G_PRE(2, 3, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
	BLAKE_G_PRE(4, 5, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
	BLAKE_G_PRE(6, 7, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
	BLAKE_G_PRE(8, 9, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
	BLAKE_G_PRE(10, 11, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
	BLAKE_G_PRE(12, 13, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
	BLAKE_G_PRE(14, 15, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);
	// { 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	BLAKE_G_PRE(14, 10, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
	BLAKE_G_PRE(4, 8, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
	BLAKE_G_PRE(9, 15, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
	BLAKE_G_PRE(13, 6, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
	BLAKE_G_PRE(1, 12, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
	BLAKE_G_PRE(0, 2, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
	BLAKE_G_PRE(11, 7, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
	BLAKE_G_PRE(5, 3, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);
	// { 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	BLAKE_G_PRE(11, 8, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
	BLAKE_G_PRE(12, 0, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
	BLAKE_G_PRE(5, 2, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
	BLAKE_G_PRE(15, 13, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
	BLAKE_G_PRE(10, 14, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
	BLAKE_G_PRE(3, 6, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
	BLAKE_G_PRE(7, 1, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
	BLAKE_G_PRE(9, 4, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);
	// { 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	BLAKE_G_PRE(7, 9, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
	BLAKE_G_PRE(3, 1, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
	BLAKE_G_PRE(13, 12, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
	BLAKE_G_PRE(11, 14, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
	BLAKE_G_PRE(2, 6, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
	BLAKE_G_PRE(5, 10, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
	BLAKE_G_PRE(4, 0, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
	BLAKE_G_PRE(15, 8, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);

	for(int x = 4; x < 10; x++)
	{
		BLAKE_G(x, 0x00, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
		BLAKE_G(x, 0x02, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
		BLAKE_G(x, 0x04, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
		BLAKE_G(x, 0x06, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
		BLAKE_G(x, 0x08, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
		BLAKE_G(x, 0x0A, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
		BLAKE_G(x, 0x0C, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
		BLAKE_G(x, 0x0E, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);
	}

	V.lo ^= V.hi ^ tmpblock;

	((uint8*)out)[0] = V.lo;
}
#endif

#if __CUDA_ARCH__ >= 500

#define BLAKE_G(idx0, idx1, a, b, c, d, key) { \
	idx = BLAKE2S_SIGMA[idx0][idx1]; a += key[idx]; \
	a += b; d = __byte_perm(d ^ a, 0, 0x1032); \
	c += d; b = rotateR(b ^ c, 12); \
	idx = BLAKE2S_SIGMA[idx0][idx1+1]; a += key[idx]; \
	a += b; d = __byte_perm(d ^ a, 0, 0x0321); \
	c += d; b = rotateR(b ^ c, 7); \
}

#define BLAKE(a, b, c, d, key1,key2) { \
	a += key1; \
	a += b; d = __byte_perm(d ^ a, 0, 0x1032); \
	c += d; b = rotateR(b ^ c, 12); \
	a += key2; \
	a += b; d = __byte_perm(d ^ a, 0, 0x0321); \
	c += d; b = rotateR(b ^ c, 7); \
}

#define BLAKE_G_PRE(idx0,idx1, a, b, c, d, key) { \
	a += key[idx0]; \
	a += b; d = __byte_perm(d ^ a, 0, 0x1032); \
	c += d; b = rotateR(b ^ c, 12); \
	a += key[idx1]; \
	a += b; d = __byte_perm(d ^ a, 0, 0x0321); \
	c += d; b = rotateR(b ^ c, 7); \
}

#define BLAKE_G_PRE0(idx0,idx1, a, b, c, d, key) { \
	a += b; d = __byte_perm(d ^ a, 0, 0x1032); \
	c += d; b = rotateR(b ^ c, 12); \
	a += b; d = __byte_perm(d^a, 0, 0x0321); \
	c += d; b = rotateR(b^c, 7); \
}

#define BLAKE_G_PRE1(idx0,idx1, a, b, c, d, key) { \
	a += key[idx0]; \
	a += b; d = __byte_perm(d ^ a, 0, 0x1032); \
	c += d; b = rotateR(b^c, 12); \
	a += b; d = __byte_perm(d ^ a, 0, 0x0321); \
	c += d; b = rotateR(b ^ c, 7); \
}

#define BLAKE_G_PRE2(idx0,idx1, a, b, c, d, key) { \
	a += b; d = __byte_perm(d ^ a, 0, 0x1032); \
	c += d; b = rotateR(b ^ c, 12); \
	a += key[idx1]; \
	a += b; d = __byte_perm(d ^ a, 0, 0x0321); \
	c += d; b = rotateR(b ^ c, 7); \
}

static __forceinline__ __device__
void Blake2S_v2(uint32_t *out, const uint32_t* __restrict__  inout, const  uint32_t * __restrict__ TheKey)
{
	uint16 V;
	uint8 tmpblock;

	V.hi = BLAKE2S_IV_Vec;
	V.lo = BLAKE2S_IV_Vec;
	V.lo.s0 ^= 0x01012020;

	// Copy input block for later
	tmpblock = V.lo;

	V.hi.s4 ^= BLAKE2S_BLOCK_SIZE;

	//	{ 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	BLAKE_G_PRE(0, 1, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(2, 3, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(4, 5, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE(6, 7, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE0(8, 9, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE0(10, 11, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE0(12, 13, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE0(14, 15, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	// { 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	BLAKE_G_PRE0(14, 10, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE1(4, 8, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE0(9, 15, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE2(13, 6, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE1(1, 12, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(0, 2, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE2(11, 7, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE(5, 3, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	// { 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	BLAKE_G_PRE0(11, 8, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE2(12, 0, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(5, 2, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE0(15, 13, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE0(10, 14, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(3, 6, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(7, 1, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE2(9, 4, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	// { 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	BLAKE_G_PRE1(7, 9, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(3, 1, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE0(13, 12, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE0(11, 14, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(2, 6, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE1(5, 10, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(4, 0, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE0(15, 8, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	// { 9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13 },
	BLAKE_G_PRE2(9, 0, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE(5, 7, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(2, 4, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE0(10, 15, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE2(14, 1, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE0(11, 12, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE1(6, 8, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE1(3, 13, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	// { 2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9 },
	BLAKE_G_PRE1(2, 12, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE1(6, 10, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE1(0, 11, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE2(8, 3, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE1(4, 13, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(7, 5, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE0(15, 14, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE1(1, 9, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	// { 12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11 },
	BLAKE_G_PRE2(12, 5, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE1(1, 15, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE0(14, 13, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE1(4, 10, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(0, 7, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE(6, 3, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE2(9, 2, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE0(8, 11, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	// { 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10 },
	BLAKE_G_PRE0(13, 11, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE1(7, 14, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE2(12, 1, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE1(3, 9, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE(5, 0, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE2(15, 4, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE2(8, 6, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE(2, 10, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	// { 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5 },
	BLAKE_G_PRE1(6, 15, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE0(14, 9, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE2(11, 3, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE1(0, 8, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE2(12, 2, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE2(13, 7, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE(1, 4, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE2(10, 5, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);
	// { 10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0 },
	BLAKE_G_PRE2(10, 2, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, TheKey);
	BLAKE_G_PRE2(8, 4, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, TheKey);
	BLAKE_G_PRE(7, 6, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, TheKey);
	BLAKE_G_PRE(1, 5, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, TheKey);
	BLAKE_G_PRE0(15, 11, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, TheKey);
	BLAKE_G_PRE0(9, 14, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, TheKey);
	BLAKE_G_PRE1(3, 12, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, TheKey);
	BLAKE_G_PRE2(13, 0, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, TheKey);

	V.lo ^= V.hi;
	V.lo ^= tmpblock;

	V.hi = BLAKE2S_IV_Vec;
	tmpblock = V.lo;

	V.hi.s4 ^= 128;
	V.hi.s6 = ~V.hi.s6;

	// { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15 },
	BLAKE_G_PRE(0, 1, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
	BLAKE_G_PRE(2, 3, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
	BLAKE_G_PRE(4, 5, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
	BLAKE_G_PRE(6, 7, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
	BLAKE_G_PRE(8, 9, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
	BLAKE_G_PRE(10, 11, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
	BLAKE_G_PRE(12, 13, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
	BLAKE_G_PRE(14, 15, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);
	// { 14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3 },
	BLAKE_G_PRE(14, 10, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
	BLAKE_G_PRE(4, 8, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
	BLAKE_G_PRE(9, 15, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
	BLAKE_G_PRE(13, 6, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
	BLAKE_G_PRE(1, 12, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
	BLAKE_G_PRE(0, 2, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
	BLAKE_G_PRE(11, 7, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
	BLAKE_G_PRE(5, 3, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);
	// { 11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4 },
	BLAKE_G_PRE(11, 8, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
	BLAKE_G_PRE(12, 0, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
	BLAKE_G_PRE(5, 2, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
	BLAKE_G_PRE(15, 13, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
	BLAKE_G_PRE(10, 14, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
	BLAKE_G_PRE(3, 6, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
	BLAKE_G_PRE(7, 1, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
	BLAKE_G_PRE(9, 4, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);
	// { 7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8 },
	BLAKE_G_PRE(7, 9, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
	BLAKE_G_PRE(3, 1, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
	BLAKE_G_PRE(13, 12, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
	BLAKE_G_PRE(11, 14, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
	BLAKE_G_PRE(2, 6, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
	BLAKE_G_PRE(5, 10, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
	BLAKE_G_PRE(4, 0, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
	BLAKE_G_PRE(15, 8, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);

	BLAKE(V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout[9], inout[0]);
	BLAKE(V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout[5], inout[7]);
	BLAKE(V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout[2], inout[4]);
	BLAKE(V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout[10], inout[15]);
	BLAKE(V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout[14], inout[1]);
	BLAKE(V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout[11], inout[12]);
	BLAKE(V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout[6], inout[8]);
	BLAKE(V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout[3], inout[13]);

	BLAKE(V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout[2], inout[12]);
	BLAKE(V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout[6], inout[10]);
	BLAKE(V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout[0], inout[11]);
	BLAKE(V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout[8], inout[3]);
	BLAKE(V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout[4], inout[13]);
	BLAKE(V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout[7], inout[5]);
	BLAKE(V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout[15], inout[14]);
	BLAKE(V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout[1], inout[9]);

	BLAKE(V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout[12], inout[5]);
	BLAKE(V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout[1], inout[15]);
	BLAKE(V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout[14], inout[13]);
	BLAKE(V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout[4], inout[10]);
	BLAKE(V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout[0], inout[7]);
	BLAKE(V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout[6], inout[3]);
	BLAKE(V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout[9], inout[2]);
	BLAKE(V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout[8], inout[11]);
	// 13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10,
	BLAKE(V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout[13], inout[11]);
	BLAKE(V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout[7], inout[14]);
	BLAKE(V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout[12], inout[1]);
	BLAKE(V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout[3], inout[9]);
	BLAKE(V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout[5], inout[0]);
	BLAKE(V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout[15], inout[4]);
	BLAKE(V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout[8], inout[6]);
	BLAKE(V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout[2], inout[10]);
	// 6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5,
	BLAKE(V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout[6], inout[15]);
	BLAKE(V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout[14], inout[9]);
	BLAKE(V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout[11], inout[3]);
	BLAKE(V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout[0], inout[8]);
	BLAKE(V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout[12], inout[2]);
	BLAKE(V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout[13], inout[7]);
	BLAKE(V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout[1], inout[4]);
	BLAKE(V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout[10], inout[5]);
	// 10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0,
	BLAKE(V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout[10], inout[2]);
	BLAKE(V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout[8], inout[4]);
	BLAKE(V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout[7], inout[6]);
	BLAKE(V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout[1], inout[5]);
	BLAKE(V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout[15], inout[11]);
	BLAKE(V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout[9], inout[14]);
	BLAKE(V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout[3], inout[12]);
	BLAKE(V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout[13], inout[0]);

	V.lo ^= V.hi;
	V.lo ^= tmpblock;

	((uint8*)out)[0] = V.lo;
}

#endif /* __CUDA_ARCH__ >= 500 */

#define SALSA_CORE(state) { \
	uint32_t t; \
	SALSA(state.x, state.y, state.z, state.w); \
	WarpShuffle3(state.y, state.z, state.w,state.y, state.z, state.w, threadIdx.x + 3, threadIdx.x + 2, threadIdx.x + 1,4); \
	SALSA(state.x, state.w, state.z, state.y); \
	WarpShuffle3(state.y, state.z, state.w,state.y, state.z, state.w, threadIdx.x + 1, threadIdx.x + 2, threadIdx.x + 3,4); \
}

#define CHACHA_CORE_PARALLEL(state)	{ \
	CHACHA_STEP(state.x, state.y, state.z, state.w); \
	WarpShuffle3(state.y, state.z, state.w,state.y, state.z, state.w, threadIdx.x + 1, threadIdx.x + 2, threadIdx.x + 3,4); \
	CHACHA_STEP(state.x, state.y, state.z, state.w); \
	WarpShuffle3(state.y, state.z, state.w, state.y, state.z, state.w, threadIdx.x + 3, threadIdx.x + 2, threadIdx.x + 1,4); \
}

static __forceinline__ __device__
uint4 salsa_small_scalar_rnd(const uint4 X)
{
	uint4 state = X;

#pragma nounroll
	for(int i = 0; i < 10; i++)
	{
		SALSA_CORE(state);
	}

	return (X + state);
}

static __device__ __forceinline__
uint4 chacha_small_parallel_rnd(const uint4 X)
{
	uint4 state = X;

#pragma nounroll
	for(int i = 0; i < 10; i++)
	{
		CHACHA_CORE_PARALLEL(state);
	}
	return (X + state);
}

static __device__ __forceinline__
void neoscrypt_chacha(uint4 XV[4])
{
	uint4 temp;

	XV[0] = chacha_small_parallel_rnd(XV[0] ^ XV[3]);
	temp = chacha_small_parallel_rnd(XV[1] ^ XV[0]);
	XV[1] = chacha_small_parallel_rnd(XV[2] ^ temp);
	XV[3] = chacha_small_parallel_rnd(XV[3] ^ XV[1]);
	XV[2] = temp;
}

static __device__ __forceinline__
void neoscrypt_salsa(uint4 XV[4])
{
	uint4 temp;

	XV[0] = salsa_small_scalar_rnd(XV[0] ^ XV[3]);
	temp = salsa_small_scalar_rnd(XV[1] ^ XV[0]);
	XV[1] = salsa_small_scalar_rnd(XV[2] ^ temp);
	XV[3] = salsa_small_scalar_rnd(XV[3] ^ XV[1]);
	XV[2] = temp;
}


#if __CUDA_ARCH__ < 500
static __forceinline__ __device__
void fastkdf256_v1(const uint32_t thread, const uint32_t nonce, uint32_t* const s_data)
{
	uint2x4 output[8];
	uint32_t* B = (uint32_t*)&s_data[threadIdx.x * 64U];
	uint32_t qbuf, rbuf, bitbuf;
	uint32_t input[BLAKE2S_BLOCK_SIZE / 4];
	uint32_t key[BLAKE2S_BLOCK_SIZE / 4] = {0};

	const uint32_t data18 = c_data[18];
	const uint32_t data20 = c_data[0];

	((uintx64*)(B))[0] = ((uintx64*)c_data)[0];
	((uint32_t*)B)[19] = nonce;
	((uint32_t*)B)[39] = nonce;
	((uint32_t*)B)[59] = nonce;
	__syncthreads();

	((uint816*)input)[0] = ((uint816*)input_init)[0];
	((uint4x2*)key)[0] = ((uint4x2*)key_init)[0];

#pragma unroll 1
	for(int i = 0; i < 31; i++)
	{
		uint32_t bufidx = 0;
#pragma unroll
		for(int x = 0; x < BLAKE2S_OUT_SIZE / 4; ++x)
		{
			uint32_t bufhelper = (input[x] & 0x00ff00ff) + ((input[x] & 0xff00ff00) >> 8);
			bufhelper = bufhelper + (bufhelper >> 16);
			bufidx += bufhelper;
		}
		bufidx &= 0x000000ff;
		qbuf = bufidx >> 2;
		rbuf = bufidx & 3;
		bitbuf = rbuf << 3;

		uint32_t shifted[9];
		shift256R4(shifted, ((uint8*)input)[0], bitbuf);

		uint32_t temp[9];
		//#pragma unroll
		for(int k = 0; k < 9; k++)
		{
			uint32_t indice = (k + qbuf) & 0x3f;
			temp[k] = B[indice] ^ shifted[k];
			B[indice] = temp[k];
		}
		__syncthreads();

		uint32_t a = c_data[qbuf & 0x3f], b;
		//#pragma unroll
		for (int k = 0; k<16; k += 2)
		{
			b = s_data[(qbuf + k + 1) & 0x0000003f];
			input[k] = __funnelshift_r(a, b, bitbuf);
			a = s_data[(qbuf + k + 2) & 0x0000003f];
			input[k + 1] = __funnelshift_r(b, a, bitbuf);
		}

		const uint32_t noncepos = 19 - qbuf % 20;
		if (noncepos <= 16 && qbuf<60)
		{
			if (noncepos != 0)
				input[noncepos - 1] = __funnelshift_r(data18, nonce, bitbuf);
			if (noncepos != 16)
				input[noncepos] = __funnelshift_r(nonce, data20, bitbuf);
		}

		for (int k = 0; k<8; k++)
			key[k] = __funnelshift_r(temp[k], temp[k + 1], bitbuf);

		Blake2S(input, input, key);
	}

	uint32_t bufidx = 0;
#pragma unroll
	for(int x = 0; x < BLAKE2S_OUT_SIZE / 4; ++x)
	{
		uint32_t bufhelper = (input[x] & 0x00ff00ff) + ((input[x] & 0xff00ff00) >> 8);
		bufhelper = bufhelper + (bufhelper >> 16);
		bufidx += bufhelper;
	}
	bufidx &= 0x000000ff;
	qbuf = bufidx >> 2;
	rbuf = bufidx & 3;
	bitbuf = rbuf << 3;

	for(int i = 0; i<64; i++)
		((uint32_t*)output)[i] = __funnelshift_r(B[(qbuf + i) & 0x3f], B[(qbuf + i + 1) & 0x3f4], bitbuf);

	((ulonglong4*)output)[0] ^= ((ulonglong4*)input)[0];
	((uintx64*)output)[0] ^= ((uintx64*)c_data)[0];
	((uint32_t*)output)[19] ^= nonce;
	((uint32_t*)output)[39] ^= nonce;
	((uint32_t*)output)[59] ^= nonce;

	for(int i = 0; i<8; i++)
		(Input + 8U * thread)[i] = output[i];
}
#endif

#if __CUDA_ARCH__ >= 500
static __forceinline__ __device__
void fastkdf256_v2(const uint32_t thread, const uint32_t nonce, uint32_t* const s_data)
{
	const uint32_t data18 = c_data[18];
	const uint32_t data20 = c_data[0];
	uint32_t input[16];
	uint32_t key[16] = {0};
	uint32_t qbuf, rbuf, bitbuf;

	uint32_t* B = &s_data[threadIdx.x * 64U];
	((uintx64*)(B))[0] = ((uintx64*)c_data)[0];

	B[19] = nonce;
	B[39] = nonce;
	B[59] = nonce;
	__syncthreads();
	{
		uint32_t bufidx = 0;
#pragma unroll
		for(int x = 0; x < BLAKE2S_OUT_SIZE / 4; ++x)
		{
			uint32_t bufhelper = (input_init[x] & 0x00ff00ff) + ((input_init[x] & 0xff00ff00) >> 8);
			bufhelper = bufhelper + (bufhelper >> 16);
			bufidx += bufhelper;
		}
		bufidx &= 0x000000ff;
		rbuf = bufidx & 3;
		qbuf = bufidx >> 2;
		bitbuf = rbuf << 3;

		uint32_t temp[9];

		uint32_t shifted;
		uint32_t shift = 32U - bitbuf;
		shifted = input_init[0] << bitbuf;
		temp[0] = B[(0 + qbuf) & 0x3f] ^ shifted;
		shifted = __funnelshift_r(input_init[0], input_init[1], shift);
		temp[1] = B[(1 + qbuf) & 0x3f] ^ shifted;
		shifted = __funnelshift_r(input_init[1], input_init[2], shift);
		temp[2] = B[(2 + qbuf) & 0x3f] ^ shifted;
		shifted = __funnelshift_r(input_init[2], input_init[3], shift);
		temp[3] = B[(3 + qbuf) & 0x3f] ^ shifted;
		shifted = __funnelshift_r(input_init[3], input_init[4], shift);
		temp[4] = B[(4 + qbuf) & 0x3f] ^ shifted;
		shifted = __funnelshift_r(input_init[4], input_init[5], shift);
		temp[5] = B[(5 + qbuf) & 0x3f] ^ shifted;
		shifted = __funnelshift_r(input_init[5], input_init[6], shift);
		temp[6] = B[(6 + qbuf) & 0x3f] ^ shifted;
		shifted = __funnelshift_r(input_init[6], input_init[7], shift);
		temp[7] = B[(7 + qbuf) & 0x3f] ^ shifted;
		shifted = input_init[7] >> shift;
		temp[8] = B[(8 + qbuf) & 0x3f] ^ shifted;

		uint32_t a = c_data[qbuf & 0x3f], b;

#pragma unroll
		for (int k = 0; k<16; k += 2)
		{
			b = s_data[(qbuf + k + 1) & 0x0000003f];
			input[k] = __funnelshift_r(a, b, bitbuf);
			a = s_data[(qbuf + k + 2) & 0x0000003f];
			input[k + 1] = __funnelshift_r(b, a, bitbuf);
		}

		const uint32_t noncepos = 19 - qbuf % 20;
		if (noncepos <= 16 && qbuf<60)
		{
			if (noncepos != 0)
				input[noncepos - 1] = __funnelshift_r(data18, nonce, bitbuf);
			if (noncepos != 16)
				input[noncepos] = __funnelshift_r(nonce, data20, bitbuf);
		}

		key[0] = __funnelshift_r(temp[0], temp[1], bitbuf);
		key[1] = __funnelshift_r(temp[1], temp[2], bitbuf);
		key[2] = __funnelshift_r(temp[2], temp[3], bitbuf);
		key[3] = __funnelshift_r(temp[3], temp[4], bitbuf);
		key[4] = __funnelshift_r(temp[4], temp[5], bitbuf);
		key[5] = __funnelshift_r(temp[5], temp[6], bitbuf);
		key[6] = __funnelshift_r(temp[6], temp[7], bitbuf);
		key[7] = __funnelshift_r(temp[7], temp[8], bitbuf);
		
		Blake2S_v2(input, input, key);

#pragma unroll
		for(int k = 0; k < 9; k++)
			B[(k + qbuf) & 0x3f] = temp[k];
		__syncthreads();
	}

	for(int i = 1; i < 31; i++)
	{
		uint32_t bufidx = 0;
#pragma unroll
		for(int x = 0; x < BLAKE2S_OUT_SIZE / 4; ++x)
		{
			uint32_t bufhelper = (input[x] & 0x00ff00ff) + ((input[x] & 0xff00ff00) >> 8);
			bufhelper = bufhelper + (bufhelper >> 16);
			bufidx += bufhelper;
		}
		bufidx &= 0x000000ff;
		qbuf = bufidx >> 2;
		rbuf = bufidx & 3;
		bitbuf = rbuf << 3;

		uint32_t temp[9];

		uint32_t shifted;
		uint32_t shift = 32U - bitbuf;
		shifted = input[0] << bitbuf;
		temp[0] = B[(0 + qbuf) & 0x3f] ^ shifted;
		shifted = __funnelshift_r(input[0], input[1], shift);
		temp[1] = B[(1 + qbuf) & 0x3f] ^ shifted;
		shifted = __funnelshift_r(input[1], input[2], shift);
		temp[2] = B[(2 + qbuf) & 0x3f] ^ shifted;
		shifted = __funnelshift_r(input[2], input[3], shift);
		temp[3] = B[(3 + qbuf) & 0x3f] ^ shifted;
		shifted = __funnelshift_r(input[3], input[4], shift);
		temp[4] = B[(4 + qbuf) & 0x3f] ^ shifted;
		shifted = __funnelshift_r(input[4], input[5], shift);
		temp[5] = B[(5 + qbuf) & 0x3f] ^ shifted;
		shifted = __funnelshift_r(input[5], input[6], shift);
		temp[6] = B[(6 + qbuf) & 0x3f] ^ shifted;
		shifted = __funnelshift_r(input[6], input[7], shift);
		temp[7] = B[(7 + qbuf) & 0x3f] ^ shifted;
		shifted = input[7] >> shift;
		temp[8] = B[(8 + qbuf) & 0x3f] ^ shifted;

		uint32_t a = c_data[qbuf & 0x3f], b;

#pragma unroll
		for (int k = 0; k<16; k += 2)
		{
			b = s_data[(qbuf + k + 1) & 0x0000003f];
			input[k] = __funnelshift_r(a, b, bitbuf);
			a = s_data[(qbuf + k + 2) & 0x0000003f];
			input[k + 1] = __funnelshift_r(b, a, bitbuf);
		}

		const uint32_t noncepos = 19 - qbuf % 20;
		if (noncepos <= 16 && qbuf<60)
		{
			if (noncepos != 0)
				input[noncepos - 1] = __funnelshift_r(data18, nonce, bitbuf);
			if (noncepos != 16)
				input[noncepos] = __funnelshift_r(nonce, data20, bitbuf);
		}

		key[0] = __funnelshift_r(temp[0], temp[1], bitbuf);
		key[1] = __funnelshift_r(temp[1], temp[2], bitbuf);
		key[2] = __funnelshift_r(temp[2], temp[3], bitbuf);
		key[3] = __funnelshift_r(temp[3], temp[4], bitbuf);
		key[4] = __funnelshift_r(temp[4], temp[5], bitbuf);
		key[5] = __funnelshift_r(temp[5], temp[6], bitbuf);
		key[6] = __funnelshift_r(temp[6], temp[7], bitbuf);
		key[7] = __funnelshift_r(temp[7], temp[8], bitbuf);

		Blake2S_v2(input, input, key);

#pragma unroll
		for(int k = 0; k < 9; k++)
			B[(k + qbuf) & 0x3f] = temp[k];
		__syncthreads();
	}

	{
		uint32_t bufidx = 0;
#pragma unroll
		for(int x = 0; x < BLAKE2S_OUT_SIZE / 4; ++x)
		{
			uint32_t bufhelper = (input[x] & 0x00ff00ff) + ((input[x] & 0xff00ff00) >> 8);
			bufhelper = bufhelper + (bufhelper >> 16);
			bufidx += bufhelper;
		}
		bufidx &= 0x000000ff;
		qbuf = bufidx >> 2;
		rbuf = bufidx & 3;
		bitbuf = rbuf << 3;
	}

	uint2x4 output[8];
	for(int i = 0; i<64; i++)
	{
		const uint32_t a = (qbuf + i) & 0x3f, b = (qbuf + i + 1) & 0x3f;
		((uint32_t*)output)[i] = __funnelshift_r(B[a], B[b], bitbuf);
	}

	output[0] ^= ((uint2x4*)input)[0];
#pragma unroll
	for(int i = 0; i<8; i++)
		output[i] ^= ((uint2x4*)c_data)[i];

	((uint32_t*)output)[19] ^= nonce;
	((uint32_t*)output)[39] ^= nonce;
	((uint32_t*)output)[59] ^= nonce;
	((ulonglong16 *)(Input + 8U * thread))[0] = ((ulonglong16*)output)[0];
}
#endif

#if __CUDA_ARCH__ < 500
static __forceinline__ __device__
uint32_t fastkdf32_v1(uint32_t thread, const uint32_t nonce, uint32_t* const salt, uint32_t* const s_data)
{
	const uint32_t cdata7 = c_data[7];
	const uint32_t data18 = c_data[18];
	const uint32_t data20 = c_data[0];

	uint32_t* B0 = (uint32_t*)&s_data[threadIdx.x * 64U];
	((uintx64*)B0)[0] = ((uintx64*)salt)[0];
	__syncthreads();

	uint32_t input[BLAKE2S_BLOCK_SIZE / 4];
	((uint816*)input)[0] = ((uint816*)c_data)[0];

	uint32_t key[BLAKE2S_BLOCK_SIZE / 4];
	((uint4x2*)key)[0] = ((uint4x2*)salt)[0];
	((uint4*)key)[2] = make_uint4(0, 0, 0, 0);
	((uint4*)key)[3] = make_uint4(0, 0, 0, 0);

	uint32_t qbuf, rbuf, bitbuf;
	uint32_t temp[9];

#pragma nounroll
	for(int i = 0; i < 31; i++)
	{
		Blake2S(input, input, key);

		uint32_t bufidx = 0;
#pragma unroll
		for(int x = 0; x < BLAKE2S_OUT_SIZE / 4; ++x)
		{
			uint32_t bufhelper = (input[x] & 0x00ff00ff) + ((input[x] & 0xff00ff00) >> 8);
			bufhelper = bufhelper + (bufhelper >> 16);
			bufidx += bufhelper;
		}
		bufidx &= 0x000000ff;
		qbuf = bufidx >> 2;
		rbuf = bufidx & 3;
		bitbuf = rbuf << 3;
		uint32_t shifted[9];

		shift256R4(shifted, ((uint8*)input)[0], bitbuf);

		for(int k = 0; k < 9; k++)
		{
			temp[k] = B0[(k + qbuf) & 0x3f];
		}

		((uint2x4*)temp)[0] ^= ((uint2x4*)shifted)[0];
		temp[8] ^= shifted[8];

		uint32_t a = c_data[qbuf & 0x3f], b;
		//#pragma unroll
		for (int k = 0; k<16; k += 2)
		{
			b = s_data[(qbuf + k + 1) & 0x0000003f];
			input[k] = __funnelshift_r(a, b, bitbuf);
			a = s_data[(qbuf + k + 2) & 0x0000003f];
			input[k + 1] = __funnelshift_r(b, a, bitbuf);
		}

		const uint32_t noncepos = 19 - qbuf % 20;
		if (noncepos <= 16 && qbuf<60)
		{
			if (noncepos != 0)
				input[noncepos - 1] = __funnelshift_r(data18, nonce, bitbuf);
			if (noncepos != 16)
				input[noncepos] = __funnelshift_r(nonce, data20, bitbuf);
		}

		key[0] = __funnelshift_r(temp[0], temp[1], bitbuf);
		key[1] = __funnelshift_r(temp[1], temp[2], bitbuf);
		key[2] = __funnelshift_r(temp[2], temp[3], bitbuf);
		key[3] = __funnelshift_r(temp[3], temp[4], bitbuf);
		key[4] = __funnelshift_r(temp[4], temp[5], bitbuf);
		key[5] = __funnelshift_r(temp[5], temp[6], bitbuf);
		key[6] = __funnelshift_r(temp[6], temp[7], bitbuf);
		key[7] = __funnelshift_r(temp[7], temp[8], bitbuf);

		for(int k = 0; k < 9; k++)
		{
			B0[(k + qbuf) & 0x3f] = temp[k];
		}
		__syncthreads();
	}

	Blake2S(input, input, key);

	uint32_t bufidx = 0;
#pragma unroll
	for(int x = 0; x < BLAKE2S_OUT_SIZE / 4; ++x)
	{
		uint32_t bufhelper = (input[x] & 0x00ff00ff) + ((input[x] & 0xff00ff00) >> 8);
		bufhelper = bufhelper + (bufhelper >> 16);
		bufidx += bufhelper;
	}
	bufidx &= 0x000000ff;
	qbuf = bufidx >> 2;
	rbuf = bufidx & 3;
	bitbuf = rbuf << 3;

	for(int k = 7; k < 9; k++)
	{
		temp[k] = B0[(k + qbuf) & 0x3f];
	}

	uint32_t output;
	output = __funnelshift_r(temp[7], temp[8], bitbuf);

	output ^= input[7] ^ cdata7;
	return output;
}
#endif

#if __CUDA_ARCH__ >= 500
static __forceinline__ __device__
uint32_t fastkdf32_v3(uint32_t thread, const uint32_t nonce, uint32_t* const salt, uint32_t* const s_data)
{
	const uint32_t cdata7 = c_data[7];
	const uint32_t data18 = c_data[18];
	const uint32_t data20 = c_data[0];

	uint32_t* B0 = (uint32_t*)&s_data[threadIdx.x * 64U];
	((uintx64*)B0)[0] = ((uintx64*)salt)[0];
	__syncthreads();

	uint32_t input[BLAKE2S_BLOCK_SIZE / 4];
	((uint816*)input)[0] = ((uint816*)c_data)[0];

	uint32_t key[BLAKE2S_BLOCK_SIZE / 4];
	((uint4x2*)key)[0] = ((uint4x2*)salt)[0];
	((uint4*)key)[2] = make_uint4(0, 0, 0, 0);
	((uint4*)key)[3] = make_uint4(0, 0, 0, 0);

	uint32_t qbuf, rbuf, bitbuf;
	uint32_t temp[9];

#pragma nounroll
	for(int i = 0; i < 31; i++)
	{
		Blake2S_v2(input, input, key);

		uint32_t bufidx = 0;
#pragma unroll
		for(int x = 0; x < BLAKE2S_OUT_SIZE / 4; ++x)
		{
			uint32_t bufhelper = (input[x] & 0x00ff00ff) + ((input[x] & 0xff00ff00) >> 8);
			bufhelper = bufhelper + (bufhelper >> 16);
			bufidx += bufhelper;
		}
		bufidx &= 0x000000ff;
		qbuf = bufidx >> 2;
		rbuf = bufidx & 3;
		bitbuf = rbuf << 3;

		uint32_t shifted;
		uint32_t shift = 32U - bitbuf;
		shifted = input[0] << bitbuf;
		temp[0] = B0[(0 + qbuf) & 0x3f] ^ shifted;
		shifted = __funnelshift_r(input[0], input[1], shift);
		temp[1] = B0[(1 + qbuf) & 0x3f] ^ shifted;
		shifted = __funnelshift_r(input[1], input[2], shift);
		temp[2] = B0[(2 + qbuf) & 0x3f] ^ shifted;
		shifted = __funnelshift_r(input[2], input[3], shift);
		temp[3] = B0[(3 + qbuf) & 0x3f] ^ shifted;
		shifted = __funnelshift_r(input[3], input[4], shift);
		temp[4] = B0[(4 + qbuf) & 0x3f] ^ shifted;
		shifted = __funnelshift_r(input[4], input[5], shift);
		temp[5] = B0[(5 + qbuf) & 0x3f] ^ shifted;
		shifted = __funnelshift_r(input[5], input[6], shift);
		temp[6] = B0[(6 + qbuf) & 0x3f] ^ shifted;
		shifted = __funnelshift_r(input[6], input[7], shift);
		temp[7] = B0[(7 + qbuf) & 0x3f] ^ shifted;
		shifted = input[7] >> shift;
		temp[8] = B0[(8 + qbuf) & 0x3f] ^ shifted;
		
		uint32_t a = c_data[qbuf & 0x3f], b;
#pragma unroll
		for (int k = 0; k<16; k += 2)
		{
			b = s_data[(qbuf + k + 1) & 0x0000003f];
			input[k] = __funnelshift_r(a, b, bitbuf);
			a = s_data[(qbuf + k + 2) & 0x0000003f];
			input[k + 1] = __funnelshift_r(b, a, bitbuf);
		}

		const uint32_t noncepos = 19 - qbuf % 20;
		if (noncepos <= 16 && qbuf<60)
		{
			if (noncepos != 0)
				input[noncepos - 1] = __funnelshift_r(data18, nonce, bitbuf);
			if (noncepos != 16)
				input[noncepos] = __funnelshift_r(nonce, data20, bitbuf);
		}

		key[0] = __funnelshift_r(temp[0], temp[1], bitbuf);
		key[1] = __funnelshift_r(temp[1], temp[2], bitbuf);
		key[2] = __funnelshift_r(temp[2], temp[3], bitbuf);
		key[3] = __funnelshift_r(temp[3], temp[4], bitbuf);
		key[4] = __funnelshift_r(temp[4], temp[5], bitbuf);
		key[5] = __funnelshift_r(temp[5], temp[6], bitbuf);
		key[6] = __funnelshift_r(temp[6], temp[7], bitbuf);
		key[7] = __funnelshift_r(temp[7], temp[8], bitbuf);

#pragma unroll
		for(int k = 0; k < 9; k++)
		{
			B0[(k + qbuf) & 0x3f] = temp[k];
		}
		__syncthreads();
	}

	Blake2S_v2(input, input, key);

	uint32_t bufidx = 0;
#pragma unroll
	for(int x = 0; x < BLAKE2S_OUT_SIZE / 4; ++x)
	{
		uint32_t bufhelper = (input[x] & 0x00ff00ff) + ((input[x] & 0xff00ff00) >> 8);
		bufhelper = bufhelper + (bufhelper >> 16);
		bufidx += bufhelper;
	}
	bufidx &= 0x000000ff;
	qbuf = bufidx >> 2;
	rbuf = bufidx & 3;
	bitbuf = rbuf << 3;

	temp[7] = B0[(qbuf + 7) & 0x3f];
	temp[8] = B0[(qbuf + 8) & 0x3f];

	uint32_t output;
	output = __funnelshift_r(temp[7], temp[8], bitbuf);
	output ^= input[7] ^ cdata7;
	return output;
}
#endif


#define BLAKE_Ghost(idx0, idx1, a, b, c, d, key) { \
	idx = BLAKE2S_SIGMA_host[idx0][idx1]; a += key[idx]; \
	a += b; d = ROTR32(d ^ a, 16); \
	c += d; b = ROTR32(b ^ c, 12); \
	idx = BLAKE2S_SIGMA_host[idx0][idx1 + 1]; a += key[idx]; \
	a += b; d = ROTR32(d ^ a, 8); \
	c += d; b = ROTR32(b ^ c, 7); \
}

static void Blake2Shost(uint32_t * inout, const uint32_t * inkey)
{
	uint16 V;
	uint32_t idx;
	uint8 tmpblock;

	V.hi = BLAKE2S_IV_Vechost;
	V.lo = BLAKE2S_IV_Vechost;
	V.lo.s0 ^= 0x01012020;

	// Copy input block for later
	tmpblock = V.lo;

	V.hi.s4 ^= BLAKE2S_BLOCK_SIZE;

	for(int x = 0; x < 10; ++x)
	{
		BLAKE_Ghost(x, 0x00, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inkey);
		BLAKE_Ghost(x, 0x02, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inkey);
		BLAKE_Ghost(x, 0x04, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inkey);
		BLAKE_Ghost(x, 0x06, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inkey);
		BLAKE_Ghost(x, 0x08, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inkey);
		BLAKE_Ghost(x, 0x0A, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inkey);
		BLAKE_Ghost(x, 0x0C, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inkey);
		BLAKE_Ghost(x, 0x0E, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inkey);
	}

	V.lo ^= V.hi;
	V.lo ^= tmpblock;

	V.hi = BLAKE2S_IV_Vechost;
	tmpblock = V.lo;

	V.hi.s4 ^= 128;
	V.hi.s6 = ~V.hi.s6;

	for(int x = 0; x < 10; ++x)
	{
		BLAKE_Ghost(x, 0x00, V.lo.s0, V.lo.s4, V.hi.s0, V.hi.s4, inout);
		BLAKE_Ghost(x, 0x02, V.lo.s1, V.lo.s5, V.hi.s1, V.hi.s5, inout);
		BLAKE_Ghost(x, 0x04, V.lo.s2, V.lo.s6, V.hi.s2, V.hi.s6, inout);
		BLAKE_Ghost(x, 0x06, V.lo.s3, V.lo.s7, V.hi.s3, V.hi.s7, inout);
		BLAKE_Ghost(x, 0x08, V.lo.s0, V.lo.s5, V.hi.s2, V.hi.s7, inout);
		BLAKE_Ghost(x, 0x0A, V.lo.s1, V.lo.s6, V.hi.s3, V.hi.s4, inout);
		BLAKE_Ghost(x, 0x0C, V.lo.s2, V.lo.s7, V.hi.s0, V.hi.s5, inout);
		BLAKE_Ghost(x, 0x0E, V.lo.s3, V.lo.s4, V.hi.s1, V.hi.s6, inout);
	}

	V.lo ^= V.hi ^ tmpblock;

	((uint8*)inout)[0] = V.lo;
}


#define SHIFT 128U
#define TPB 32
#define TPB2 64

__global__
__launch_bounds__(TPB2, 1)
void neoscrypt_gpu_hash_start(uint32_t threads, const int stratum, const uint32_t startNonce)
{
	extern __shared__ uint32_t s_data[];

	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if(thread >= threads)
		return;
	const uint32_t ZNonce = (stratum) ? cuda_swab32(startNonce + thread) : (startNonce + thread); //freaking morons !!!

#if __CUDA_ARCH__ < 500
	fastkdf256_v1(thread, ZNonce, s_data);
#else
	fastkdf256_v2(thread, ZNonce, s_data);
#endif
}

__global__
__launch_bounds__(TPB, 1)
void neoscrypt_gpu_hash_chacha1()
{
	const uint32_t thread = (blockDim.y * blockIdx.x + threadIdx.y);
	const uint32_t threads = (gridDim.x * blockDim.y);
	const uint32_t shiftTr = 8U * thread;
	if(thread >= threads)
		return;

	uint4 X[4];
	for(int i = 0; i < 4; i++)
	{
		X[i].x = __ldg((uint32_t*)&(Input + shiftTr)[i * 2] + 0 * 4 + threadIdx.x);
		X[i].y = __ldg((uint32_t*)&(Input + shiftTr)[i * 2] + 1 * 4 + threadIdx.x);
		X[i].z = __ldg((uint32_t*)&(Input + shiftTr)[i * 2] + 2 * 4 + threadIdx.x);
		X[i].w = __ldg((uint32_t*)&(Input + shiftTr)[i * 2] + 3 * 4 + threadIdx.x);
	}

#pragma nounroll
	for(int i = 0; i < 128; i++)
	{
		ptrdiff_t offset = 8U * (thread + threads * i);
		for(int j = 0; j < 4; j++)
			((uint4*)(W + offset))[j * 4 + threadIdx.x] = X[j];
		neoscrypt_chacha(X);
	}

#pragma nounroll
	for(int t = 0; t < 128; t++)
	{
		ptrdiff_t offset = 8U * (thread + threads * (WarpShuffle(X[3].x, 0, 4) & 0x7F));
		for(int j = 0; j < 4; j++)
			X[j] ^= ((uint4*)(W + offset))[j * 4 + threadIdx.x];
		neoscrypt_chacha(X);
	}

#pragma unroll
	for(int i = 0; i < 4; i++)
	{
		*((uint32_t*)&(Tr + shiftTr)[i * 2] + 0 * 4 + threadIdx.x) = X[i].x;
		*((uint32_t*)&(Tr + shiftTr)[i * 2] + 1 * 4 + threadIdx.x) = X[i].y;
		*((uint32_t*)&(Tr + shiftTr)[i * 2] + 2 * 4 + threadIdx.x) = X[i].z;
		*((uint32_t*)&(Tr + shiftTr)[i * 2] + 3 * 4 + threadIdx.x) = X[i].w;
	}
}

__global__
__launch_bounds__(TPB, 1)
void neoscrypt_gpu_hash_salsa1()
{
	const uint32_t thread = (blockDim.y * blockIdx.x + threadIdx.y);
	const uint32_t threads = (gridDim.x * blockDim.y);
	const uint32_t shiftTr = 8U * thread;
	if(thread >= threads)
		return;

	uint4 Z[4];
	for(int i = 0; i < 4; i++)
	{
		Z[i].x = __ldg((uint32_t*)&(Input + shiftTr)[i * 2] + ((0 + threadIdx.x) & 3) * 4 + threadIdx.x);
		Z[i].y = __ldg((uint32_t*)&(Input + shiftTr)[i * 2] + ((1 + threadIdx.x) & 3) * 4 + threadIdx.x);
		Z[i].z = __ldg((uint32_t*)&(Input + shiftTr)[i * 2] + ((2 + threadIdx.x) & 3) * 4 + threadIdx.x);
		Z[i].w = __ldg((uint32_t*)&(Input + shiftTr)[i * 2] + ((3 + threadIdx.x) & 3) * 4 + threadIdx.x);
	}

#pragma nounroll
	for(int i = 0; i < 128; i++)
	{
		ptrdiff_t offset = 8U * (thread + threads * i);
		for(int j = 0; j < 4; j++)
			((uint4*)(W + offset))[j * 4 + threadIdx.x] = Z[j];
		neoscrypt_salsa(Z);
	}

#pragma nounroll
	for(int t = 0; t < 128; t++)
	{
		uint32_t offset = 8U * (thread + threads * (WarpShuffle(Z[3].x, 0, 4) & 0x7F));
		for(int j = 0; j < 4; j++)
			Z[j] ^= ((uint4*)(W + offset))[j * 4 + threadIdx.x];
		neoscrypt_salsa(Z);
	}
#pragma unroll
	for(int i = 0; i < 4; i++)
	{
		*((uint32_t*)&(Tr2 + shiftTr)[i * 2] + ((0 + threadIdx.x) & 3) * 4 + threadIdx.x) = Z[i].x;
		*((uint32_t*)&(Tr2 + shiftTr)[i * 2] + ((1 + threadIdx.x) & 3) * 4 + threadIdx.x) = Z[i].y;
		*((uint32_t*)&(Tr2 + shiftTr)[i * 2] + ((2 + threadIdx.x) & 3) * 4 + threadIdx.x) = Z[i].z;
		*((uint32_t*)&(Tr2 + shiftTr)[i * 2] + ((3 + threadIdx.x) & 3) * 4 + threadIdx.x) = Z[i].w;
	}
}

__global__
__launch_bounds__(TPB2, 8)
void neoscrypt_gpu_hash_ending(uint32_t threads, const int stratum, const uint32_t startNonce, uint32_t *resNonces)
{
	extern __shared__ uint32_t s_data[];

	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
	const uint32_t shiftTr = thread * 8U;
	const uint32_t nonce = startNonce + thread;
	const uint32_t ZNonce = (stratum) ? cuda_swab32(nonce) : nonce;
	if(thread >= threads)
		return;

	__syncthreads();

	uint2x4 Z[8];
#pragma unroll
	for(int i = 0; i<8; i++)
		Z[i] = __ldg4(&(Tr2 + shiftTr)[i]) ^ __ldg4(&(Tr + shiftTr)[i]);

#if __CUDA_ARCH__ < 500
	uint32_t outbuf = fastkdf32_v1(thread, ZNonce, (uint32_t*)Z, s_data);
#else
	uint32_t outbuf = fastkdf32_v3(thread, ZNonce, (uint32_t*)Z, s_data);
#endif

	if(outbuf <= c_target[1])
	{
		uint32_t tmp = atomicExch(resNonces, nonce);
		if(tmp != 0xffffffff)
			resNonces[1] = tmp;
	}
}

static THREAD uint32_t *hash1 = NULL;
static THREAD uint32_t *Trans1 = NULL;
static THREAD uint32_t *Trans2 = NULL; // 2 streams
static THREAD uint32_t *Trans3 = NULL; // 2 streams

__host__
void neoscrypt_init(int thr_id, uint32_t threads)
{
#if CUDART_VERSION >= 9000
	CUDA_SAFE_CALL(hipFuncSetAttribute(reinterpret_cast<const void*>(neoscrypt_gpu_hash_start), hipFuncAttributePreferredSharedMemoryCarveout, 100)); // make Titan V faster
	CUDA_SAFE_CALL(hipFuncSetAttribute(reinterpret_cast<const void*>(neoscrypt_gpu_hash_ending), hipFuncAttributePreferredSharedMemoryCarveout, 100)); // make Titan V faster
#endif
	CUDA_SAFE_CALL(hipMalloc(&d_NNonce[thr_id], 2 * sizeof(uint32_t)));
	CUDA_SAFE_CALL(hipMalloc(&hash1, 32ULL * 128 * sizeof(uint64_t) * threads));
	CUDA_SAFE_CALL(hipMalloc(&Trans1, 32ULL * sizeof(uint64_t) * threads));
	CUDA_SAFE_CALL(hipMalloc(&Trans2, 32ULL * sizeof(uint64_t) * threads));
	CUDA_SAFE_CALL(hipMalloc(&Trans3, 32ULL * sizeof(uint64_t) * threads));

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(W), &hash1, sizeof(uint2x4*), 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(Tr), &Trans1, sizeof(uint2x4*), 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(Tr2), &Trans2, sizeof(uint2x4*), 0, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(Input), &Trans3, sizeof(uint2x4*), 0, hipMemcpyHostToDevice));
}
/*
__host__
void neoscrypt_free(int thr_id)
{
	hipFree(d_NNonce[thr_id]);

	hipFree(hash1);
	hipFree(Trans1);
	hipFree(Trans2);
	hipFree(Trans3);
}
*/
__host__
void neoscrypt_hash_tpruvot(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *resNonces, bool stratum)
{
	const int threadsperblock2 = TPB2;
	dim3 grid2((threads + threadsperblock2 - 1) / threadsperblock2);
	dim3 block2(threadsperblock2);

	const int threadsperblock = TPB;
	dim3 grid3((threads * 4 + threadsperblock - 1) / threadsperblock);
	dim3 block3(4, threadsperblock >> 2);

	neoscrypt_gpu_hash_start << <grid2, block2 ,64* threadsperblock2 >> > (threads, stratum, startNounce); //fastkdf
	CUDA_SAFE_CALL(hipGetLastError());
	neoscrypt_gpu_hash_salsa1 << <grid3, block3 >> > ();
	CUDA_SAFE_CALL(hipGetLastError());
	neoscrypt_gpu_hash_chacha1 << <grid3, block3 >> > ();
	if(opt_debug)
		CUDA_SAFE_CALL(hipDeviceSynchronize());
	CUDA_SAFE_CALL(hipGetLastError());

	neoscrypt_gpu_hash_ending << <grid2, block2, 64 * threadsperblock2 >> > (threads, stratum, startNounce, d_NNonce[thr_id]); //fastkdf+end
	if(opt_debug)
		CUDA_SAFE_CALL(hipDeviceSynchronize());
	CUDA_SAFE_CALL(hipMemcpy(resNonces, d_NNonce[thr_id], 2 * sizeof(uint32_t), hipMemcpyDeviceToHost));
}

__host__
void neoscrypt_setBlockTarget_tpruvot(int thr_id, uint32_t* const pdata, uint32_t* const target)
{
	uint32_t PaddedMessage[64];
	uint32_t input[16], key[16] = {0};

	for(int i = 0; i < 19; i++)
	{
		PaddedMessage[i] = pdata[i];
		PaddedMessage[i + 20] = pdata[i];
		PaddedMessage[i + 40] = pdata[i];
	}
	for(int i = 0; i<4; i++)
		PaddedMessage[i + 60] = pdata[i];

	PaddedMessage[19] = 0;
	PaddedMessage[39] = 0;
	PaddedMessage[59] = 0;

	((uint16*)input)[0] = ((uint16*)pdata)[0];
	((uint8*)key)[0] = ((uint8*)pdata)[0];

	Blake2Shost(input, key);

	CUDA_SAFE_CALL(hipMemset(d_NNonce[thr_id], 0xff, 2 * sizeof(uint32_t)));

	hipMemcpyToSymbol(HIP_SYMBOL(input_init), input, 64, 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(key_init), key, 64, 0, hipMemcpyHostToDevice);

	hipMemcpyToSymbol(HIP_SYMBOL(c_target), &target[6], 2 * sizeof(uint32_t), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(c_data), PaddedMessage, 64 * sizeof(uint32_t), 0, hipMemcpyHostToDevice);
	CUDA_SAFE_CALL(hipGetLastError());
	if(opt_debug)
		CUDA_SAFE_CALL(hipDeviceSynchronize());
}

__global__ void get_cuda_arch_neo_tpruvot_gpu(int *d_version)
{
#ifdef __CUDA_ARCH__
	*d_version = __CUDA_ARCH__;
#endif
}

__host__ void get_cuda_arch_neo_tpruvot(int *version)
{
	int *d_version;
	hipMalloc(&d_version, sizeof(int));
	get_cuda_arch_neo_tpruvot_gpu << < 1, 1 >> > (d_version);
	hipMemcpy(version, d_version, sizeof(int), hipMemcpyDeviceToHost);
	hipFree(d_version);
}
